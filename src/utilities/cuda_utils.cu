#include "cuda_utils.cuh"

hipError_t sync()
{
	return hipDeviceSynchronize();
}

hipError_t peek()
{
	return hipPeekAtLastError();
}

hipError_t reset()
{
	return hipDeviceReset();
}

hipError_t copy
(
	void* dst,
	void* src,
	size_t bytes
)
{
	hipError_t error = hipMemcpy
	(
		dst,
		src,
		bytes,
		hipMemcpyDefault
	);

	return error;
}

__host__ __device__
void* malloc_device
(
	size_t bytes
)
{
	void* ptr;
	
	hipMalloc
	(
		&ptr,
		bytes
	);

	return ptr;
}

__host__ __device__
hipError_t free_device
(
	void* ptr
)
{
	return (nullptr != ptr) ? hipFree(ptr) : hipSuccess;
}