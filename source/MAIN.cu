#include "hip/hip_runtime.h"
// RC's suggestion to manage Intellisense
#ifdef __INTELLISENSE__
    #ifndef __HIPCC__
        #define __HIPCC__
    #endif
#endif

//-----------------Custom headers-----------------//

// Kernels
#include "generate_all_morton_codes.cuh"
#include "copy_finest_coefficients.cuh"
#include "insert_point_srcs.cuh"
#include "reinsert_point_srcs.cuh"
#include "init_sig_details.cuh"
#include "zero_details.cuh"
#include "traverse_tree_of_sig_details.cuh"
#include "find_neighbours.cuh"
#include "get_compaction_flags.cuh"
#include "load_soln_and_nghbr_coeffs.cuh"
#include "add_ghost_cells.cuh"
#include "friction_implicit.cuh"
#include "fv1_update.cuh"
#include "dg2_update.cuh"

// Kernel wrappers
#include "get_nodal_values.cuh"
#include "get_modal_values.cuh"
#include "sort_finest_scale_coeffs_z_order.cuh"
#include "get_max_scale_coeffs.cuh"
#include "preflag_topo.cuh"
#include "encoding_all.cuh"
#include "get_reg_tree.cuh"
#include "decoding_all.cuh"
#include "rev_z_order_act_idcs.cuh"
#include "rev_z_order_reals.cuh"
#include "sort_neighbours_z_order.cuh"
#include "compaction.cuh"
#include "get_dt_CFL.cuh"

// Input/output
#include "read_bound_conds.h"
#include "read_cell_size.h"
#include "read_plot_params.h"
#include "read_respath.h"
#include "read_save_interval.h"
#include "read_sim_params.h"
#include "read_solver_params.h"
#include "read_test_case.h"
#include "write_all_raster_maps.cuh"
#include "write_c_prop_data.cuh"
#include "write_mesh_info.h"
#include "write_gauge_point_data.cuh"
#include "write_soln_planar.cuh"
#include "write_soln_row_major.cuh"
#include "write_soln_vtk.cuh"

// Helper functions
#include "get_lvl_idx.cuh"
#include "preflag_details.cuh"
#include "project_assem_sol.cuh"
#include "copy_to_buf_assem_sol.cuh"

// Sorting
#include "get_sorting_indices.cuh"

//------------------------------------------------//

/*
 * SYNTHETIC TEST CASES:
 * 
 * The following test cases can be run by inputting a number between 1 and 22 in inputs.par:
 * 
 * 1:  Wet 1D c-property x direction
 * 2:  Wet 1D c-property y direction
 * 3:  Wet-dry 1D c-property x direction
 * 4:  Wet-dry 1D c-property y direction
 * 5:  Wet dam break in x direction
 * 6:  Wet dam break in y direction
 * 7:  Dry dam break in x direction
 * 8:  Dry dam break in y direction
 * 9:  Dry dam break in x direction with friction
 * 10: Dry dam break in y direction with friction
 * 11: Wet building overtopping in x direction
 * 12: Wet building overtopping in y direction
 * 13: Wet-dry building overtopping in x direction
 * 14: Wet-dry building overtopping in y direction
 * 15: Triangular dam break in x direction
 * 16: Triangular dam break in y direction
 * 17: Parabolic bowl in x direction
 * 18: Parabolic bowl in y direction
 * 19: Three cones
 * 20: Differentiable blocks
 * 21: Non-differentiable blocks
 * 22: Radial dam break
 * 
 */

/*
 * TOPOGRAPHY AND FLOW VARIABLES:
 *
 *   z : topography
 *   h : water height
 * eta : free surface height (h + z)
 *  qx : discharge in x direction
 *  qy : discharge in y direction
 * 
 */


/*
 * STENCIL FOR MULTIRESOLUTION ANALYSIS (MRA):
 *
 * Fig 2 of Kesserwani and Sharifian et al. (2020) takes the origin to be the bottom left corner.
 * However, the origin in this code is taken to be the top left corner.
 * This means that numbering of the child sub-elements, or children, is flipped vertically, giving:
 *
 * |-----|-----|
 * |  0  |  1  |
 * |-----|-----|
 * |  2  |  3  |
 * |-----|-----|
 *
 */


/*
 * ARRAY OF HIERARCHY OF GRIDS:
 * 
 * As a result of MRA, the 2D mesh, or grid, is square i.e. the mesh dimensions are equal.
 * Furthermore, instead of a single grid, there is a hierarchy of grids that are stacked on top of one another.
 * The resolution of each grid becomes methodically finer the further up in the hierarchy.
 * The top-most grid is at the finest resolution, which is dictated by the maximum refinement level 'L'.
 * It consists of 4^L non-overlapping elements, resulting in a mesh dimension of 2^L elements.
 * Generally, at a given refinement level n, there are 4^n elements, where n = 0, 1, ..., L - 1, L.
 * Hence, in the entire hierarchy, there 4^0 + 4^l + ... + 4^L = (4^(L+1) - 1) / 3 elements.
 * The hierarchy of grids is thus stored in a 1D array of length (4^(L+1) - 1) / 3.
 * Each grid is transformed into a 1D structure by mapping its elements to a z-order curve.
 * The z-order curve of a grid is obtained by calculating and then sorting the Morton codes of its elements.
 * Within the 1D array, the z-order curve of a grid at n + 1 begins after that of a grid at n ends.
 * 
 */

// ======================================================================================================== //
// =============================================MAIN PROGRAM=============================================== //
// ======================================================================================================== //

int main
(
	int    argc,
	char **argv
)
{
	// begin timing from the beginning, as input is automated
	const clock_t start = clock();

	// ================ //
	// TEST CASE SET UP //
	// ================ //

	const char* input_filename = argv[1];
	
	const int test_case = read_test_case(input_filename);
	
	char respath[255] = {'\0'};
	read_respath(input_filename, respath);

	// ================ //

	// =========================================================== //
	// INITIALISATION OF VARIABLES AND INSTANTIATION OF STRUCTURES //
	// =========================================================== //

	// Structures setting up simulation
	SolverParams     solver_params = read_solver_params(input_filename);
	SimulationParams sim_params    = read_sim_params(test_case, input_filename, solver_params);
	PlottingParams   plot_params   = read_plot_params(input_filename);
	Depths1D         bcs           = read_bound_conds(test_case);
	SaveInterval     saveint       = read_save_interval(input_filename, "saveint");
	SaveInterval     massint       = read_save_interval(input_filename, "massint");

	// Variables
	int mesh_dim      = 1 << solver_params.L;
	int interface_dim = mesh_dim + 1;

	real dx_finest = (test_case != 0) ? (sim_params.xmax - sim_params.xmin) / mesh_dim : read_cell_size(input_filename);
	real dy_finest = (test_case != 0) ? (sim_params.ymax - sim_params.ymin) / mesh_dim : read_cell_size(input_filename);
	real dt        = C(0.001);

	int num_finest_elems      = mesh_dim * mesh_dim;
	int num_blocks_finest     = get_num_blocks(num_finest_elems, THREADS_PER_BLOCK);
	int num_threads_traversal = num_finest_elems / 4;
	int num_blocks_traversal  = get_num_blocks(num_threads_traversal, THREADS_PER_BLOCK);
	int num_all_elems         = get_lvl_idx(solver_params.L + 1);
	int num_details           = get_lvl_idx(solver_params.L);
	int num_blocks_details    = get_num_blocks(num_details, THREADS_PER_BLOCK);
	int num_blocks_sol        = 0;
	int num_blocks_all        = get_num_blocks(num_all_elems, THREADS_PER_BLOCK);
	
	HierarchyIndex finest_lvl_idx = get_lvl_idx(solver_params.L);
	
	// Structures
	Maxes maxes = { C(1.0), C(1.0), C(1.0), C(1.0) };
	
	GaugePoints  gauge_points (input_filename, sim_params, dx_finest);
	Boundaries   boundaries   (input_filename, sim_params, dx_finest, test_case);
	PointSources point_sources(input_filename, sim_params, dx_finest, test_case, dt);
	
	clock_t end             = clock();
	real    run_time        = C(0.0);
	real    time_now        = C(0.0);
	bool    first_t_step    = true;
	bool    for_nghbrs      = false;
	bool    rkdg2           = false;
	float   avg_cuda_time   = 0.0f;
	int     steps           = 0;
	real    compression     = C(0.0);

	NodalValues       d_nodal_vals      (interface_dim);
	AssembledSolution d_assem_sol       (num_finest_elems, solver_params.solver_type);
	AssembledSolution d_buf_assem_sol   (num_finest_elems, solver_params.solver_type);
	AssembledSolution d_plot_assem_sol  (num_finest_elems, solver_params.solver_type);
	Neighbours        d_neighbours      (num_finest_elems, solver_params.solver_type);
	Neighbours        d_buf_neighbours  (num_finest_elems, solver_params.solver_type);
	ScaleCoefficients d_scale_coeffs    (num_all_elems,    solver_params.solver_type);
	Details           d_details         (num_details,      solver_params.solver_type);
	CompactionFlags   d_compaction_flags(num_finest_elems);
	FinestGrid        p_finest_grid     (num_finest_elems);
	
	// Bytesizes
	size_t bytes_morton  = num_finest_elems * sizeof(MortonCode);
	size_t bytes_details = num_details      * sizeof(real);
	size_t bytes_soln    = num_finest_elems * sizeof(real);

	// Arrays
	MortonCode* d_morton_codes        = (MortonCode*)malloc_device(bytes_morton);
	MortonCode* d_sorted_morton_codes = (MortonCode*)malloc_device(bytes_morton);
	MortonCode* d_indices             = (MortonCode*)malloc_device(bytes_morton);
	MortonCode* d_rev_z_order         = (MortonCode*)malloc_device(bytes_morton);
	MortonCode* d_rev_row_major       = (MortonCode*)malloc_device(bytes_morton);
	real*       d_eta_temp            = (real*)malloc_device(bytes_soln);
	real*       d_norm_details        = (real*)malloc_device(bytes_details);
	bool*       d_sig_details         = (bool*)malloc_device(num_details);
	real*       d_dt_CFL              = (real*)malloc_device(bytes_soln);
	
	bool* d_preflagged_details = preflag_details
	(
		boundaries, 
		point_sources, 
		gauge_points, 
		sim_params, 
		num_details, 
		solver_params.L, 
		test_case
	);

	// =========================================================== //

	// ================ //
	// INPUT AND OUTPUT //
	// ================ //

	write_mesh_info(sim_params, mesh_dim, respath);

	// ================ //

	/*
		
		hipEvent_t cuda_begin, cuda_end;
		hipEventCreate(&cuda_begin);
		hipEventCreate(&cuda_end);

		hipEventRecord(cuda_begin);


		hipEventRecord(cuda_end);
		hipEventSynchronize(cuda_end);

		float cuda_time = 0;
		hipEventElapsedTime(&cuda_time, cuda_begin, cuda_end);
		hipEventDestroy(cuda_begin);
		hipEventDestroy(cuda_end);

		avg_cuda_time += cuda_time;

	*/

	// ================================ //
	// PREPROCESSING BEFORE SOLVER LOOP //
	// ================================ //
	
	if (test_case != 0)
	{
		get_nodal_values
		(
			d_nodal_vals,
			dx_finest,
			dy_finest,
			bcs,
			sim_params,
			interface_dim,
			test_case
		);
	}

	CHECK_CUDA_ERROR(peek());
	CHECK_CUDA_ERROR(sync());

	get_modal_values
	(
		d_nodal_vals,
		d_buf_assem_sol,
		solver_params,
		sim_params,
		mesh_dim,
		interface_dim,
		test_case,
		input_filename
	);

	CHECK_CUDA_ERROR(peek());
	CHECK_CUDA_ERROR(sync());
	
	write_all_raster_maps
	(
		respath,
		d_buf_assem_sol,
		sim_params,
		solver_params,
		massint,
		mesh_dim,
		dx_finest,
		first_t_step
	);
	
	generate_all_morton_codes<<<num_blocks_finest, THREADS_PER_BLOCK>>>
	(
		d_morton_codes,
		d_indices,
		mesh_dim
	);

	CHECK_CUDA_ERROR(peek());
	CHECK_CUDA_ERROR(sync());

	get_sorting_indices
	(
		d_morton_codes,
		d_sorted_morton_codes,
		d_buf_assem_sol,
		d_assem_sol,
		d_indices,
		d_rev_z_order,
		d_rev_row_major,
		solver_params
	);

	CHECK_CUDA_ERROR(peek());
	CHECK_CUDA_ERROR(sync());

	sort_finest_scale_coeffs_z_order<<<num_blocks_finest, THREADS_PER_BLOCK>>>
	(
		d_buf_assem_sol,
		d_assem_sol,
		d_rev_z_order,
		solver_params
	);

	CHECK_CUDA_ERROR(peek());
	CHECK_CUDA_ERROR(sync());

	copy_finest_coefficients<<<num_blocks_finest, THREADS_PER_BLOCK>>>
	(
		d_assem_sol,
		d_scale_coeffs,
		solver_params,
		finest_lvl_idx
	);

	CHECK_CUDA_ERROR(peek());
	CHECK_CUDA_ERROR(sync());

	if (point_sources.num_srcs > 0)
	{
		insert_point_srcs<<<get_num_blocks(point_sources.num_srcs, THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>
		(
			d_assem_sol, 
			point_sources, 
			dt, 
			dx_finest
		);
	}

	CHECK_CUDA_ERROR(peek());
	CHECK_CUDA_ERROR(sync());

	init_sig_details<<<num_blocks_details, THREADS_PER_BLOCK>>> //d_sig_details[idx] = true;
	(
		d_sig_details, 
		num_details
	);

	CHECK_CUDA_ERROR(peek());
	CHECK_CUDA_ERROR(sync());

	maxes = get_max_scale_coeffs(d_assem_sol, d_eta_temp);

	CHECK_CUDA_ERROR(peek());
	CHECK_CUDA_ERROR(sync());

	preflag_topo
	(
		d_scale_coeffs, 
		d_details,  
		d_preflagged_details, 
		maxes,
		solver_params,
		sim_params,
		first_t_step
	);

	CHECK_CUDA_ERROR(peek());
	CHECK_CUDA_ERROR(sync());

	// ================================ //

	// ================ //
	// MAIN SOLVER LOOP //
	// ================ //
	
	while (time_now < sim_params.time)
	{
		time_now += dt;

		if ( (time_now - sim_params.time) > C(0.0) )
		{
			time_now -= dt;
			dt = sim_params.time - time_now;
			time_now += dt;
		}
		
		zero_details<<<num_blocks_details, THREADS_PER_BLOCK>>>
		(
			d_details,
			d_norm_details,
			num_details,
			solver_params
		);

		CHECK_CUDA_ERROR(peek());
		CHECK_CUDA_ERROR(sync());

		maxes = get_max_scale_coeffs(d_assem_sol, d_eta_temp);

		if (!first_t_step)
		{						
			reinsert_assem_sol<<<num_blocks_sol, THREADS_PER_BLOCK>>>
			(
				d_assem_sol,
				d_assem_sol.act_idcs,
				d_scale_coeffs,
				solver_params
			);
		}

		CHECK_CUDA_ERROR(peek());
		CHECK_CUDA_ERROR(sync());

		point_sources.update_all_srcs(time_now);

		if (point_sources.num_srcs > 0)
		{
			reinsert_point_srcs<<<get_num_blocks(point_sources.num_srcs, THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>
			(
				d_scale_coeffs, 
				point_sources, 
				dt, 
				dx_finest, 
				solver_params.L
			);
		}

		CHECK_CUDA_ERROR(peek());
		CHECK_CUDA_ERROR(sync());

		if (solver_params.epsilon > C(0.0) || first_t_step)
		{
		    for_nghbrs = false;
		    
		    encoding_all
		    (
		    	d_scale_coeffs,
		    	d_details,
		    	d_norm_details,
		    	d_sig_details,
		    	d_preflagged_details,
		    	maxes,
		    	solver_params,
		    	for_nghbrs
		    );
		    
		    get_reg_tree
		    (
		    	d_sig_details,
		    	solver_params
		    );
		    
		    CHECK_CUDA_ERROR(peek());
		    CHECK_CUDA_ERROR(sync());
		    
		    decoding_all // contains extra sig
		    (
		    	d_sig_details,
		    	d_norm_details,
		    	d_details,
		    	d_scale_coeffs,
		    	solver_params
		    );
		    
		    CHECK_CUDA_ERROR(peek());
		    CHECK_CUDA_ERROR(sync());
		    
		    traverse_tree_of_sig_details<<<num_blocks_traversal, THREADS_PER_BLOCK>>>
		    (
		    	d_sig_details,
		    	d_scale_coeffs,
		    	d_buf_assem_sol,
		    	num_threads_traversal,
		    	solver_params
		    );
		    
		    CHECK_CUDA_ERROR(peek());
		    CHECK_CUDA_ERROR(sync());
		    
		    rev_z_order_act_idcs<<<num_blocks_finest, THREADS_PER_BLOCK>>>
		    (
				d_rev_row_major,
		    	d_buf_assem_sol,
		    	d_assem_sol,
		    	num_finest_elems
		    );
		    
		    CHECK_CUDA_ERROR(peek());
		    CHECK_CUDA_ERROR(sync());
		    
		    find_neighbours<<<num_blocks_finest, THREADS_PER_BLOCK>>>
		    (
		    	d_assem_sol,
		    	d_neighbours,
		    	sim_params,
		    	mesh_dim
		    );
		    
		    CHECK_CUDA_ERROR(peek());
		    CHECK_CUDA_ERROR(sync());
		    
		    get_compaction_flags<<<num_blocks_finest, THREADS_PER_BLOCK>>>
		    (
		    	d_buf_assem_sol,
		    	d_compaction_flags,
		    	num_finest_elems
		    );
		    
		    CHECK_CUDA_ERROR(peek());
		    CHECK_CUDA_ERROR(sync());
		    
		    sort_neighbours_z_order<<<num_blocks_finest, THREADS_PER_BLOCK>>>
		    (
		    	d_neighbours,
		    	d_buf_neighbours,
		    	d_rev_z_order,
		    	num_finest_elems,
		    	solver_params
		    );
		    
		    CHECK_CUDA_ERROR(peek());
		    CHECK_CUDA_ERROR(sync());
		    
		    compaction
		    (
		    	d_buf_assem_sol,
		    	d_assem_sol,
		    	d_buf_neighbours,
		    	d_neighbours,
		    	d_compaction_flags,
		    	num_finest_elems,
		    	solver_params
		    );
		    
		    CHECK_CUDA_ERROR(peek());
		    CHECK_CUDA_ERROR(sync());
		}

		// GRID DIMENSIONS BASED ON ASSEMBLED SOLUTION LENGTH //

		num_blocks_sol = get_num_blocks(d_assem_sol.length, THREADS_PER_BLOCK);

		// -------------------------------------------------- //

		load_soln_and_nghbr_coeffs<<<num_blocks_sol, THREADS_PER_BLOCK>>>
		(
			d_neighbours,
			d_scale_coeffs,
			d_assem_sol,
			solver_params
		);

		CHECK_CUDA_ERROR(peek());
		CHECK_CUDA_ERROR(sync());
		
		boundaries.update_all_inlets(input_filename, time_now);

		add_ghost_cells<<<num_blocks_sol, THREADS_PER_BLOCK>>>
		(
			d_assem_sol,
			d_neighbours,
			solver_params,
			sim_params,
			boundaries,
			dt,
			dx_finest,
			test_case
		);

		CHECK_CUDA_ERROR(peek());
		CHECK_CUDA_ERROR(sync());

		if ( sim_params.manning > C(0.0) )
		{
			friction_implicit<<<num_blocks_sol, THREADS_PER_BLOCK>>>
			(
				d_assem_sol,
				d_neighbours,
				solver_params, 
				sim_params, 
				dt
			);
		}

		CHECK_CUDA_ERROR(peek());
		CHECK_CUDA_ERROR(sync());
		
		if (solver_params.solver_type == HWFV1)
		{
			fv1_update<<<num_blocks_sol, THREADS_PER_BLOCK>>>
			(
				d_neighbours,
				d_assem_sol,
				solver_params,
				sim_params,
				dx_finest,
				dy_finest,
				dt,
				d_dt_CFL
			);
		}
		else if (solver_params.solver_type == MWDG2)
		{
			copy_to_buf_assem_sol
			(
				d_assem_sol, 
				d_buf_assem_sol
			);

			rkdg2 = false;

			dg2_update<<<num_blocks_sol, THREADS_PER_BLOCK>>>
			(
				d_neighbours, 
				d_assem_sol, 
				d_buf_assem_sol, 
				solver_params, 
				sim_params, 
				dx_finest, 
				dy_finest, 
				dt, 
				test_case,
				d_dt_CFL,
				rkdg2
			);

			CHECK_CUDA_ERROR(peek());
			CHECK_CUDA_ERROR(sync());
			
			reinsert_assem_sol<<<num_blocks_sol, THREADS_PER_BLOCK>>>
			(
				d_buf_assem_sol,
				d_assem_sol.act_idcs,
				d_scale_coeffs,
				solver_params
			);

			CHECK_CUDA_ERROR(peek());
			CHECK_CUDA_ERROR(sync());
			
			for_nghbrs = true;

			encoding_all
			(
				d_scale_coeffs,
				d_details,
				d_norm_details,
				d_sig_details,
				d_preflagged_details,
				maxes,
				solver_params,
				for_nghbrs
			);

			CHECK_CUDA_ERROR(peek());
			CHECK_CUDA_ERROR(sync());

			load_soln_and_nghbr_coeffs<<<num_blocks_sol, THREADS_PER_BLOCK>>>
			(
				d_neighbours,
				d_scale_coeffs,
				d_buf_assem_sol,
				solver_params
			);

			CHECK_CUDA_ERROR(peek());
			CHECK_CUDA_ERROR(sync());

			add_ghost_cells<<<num_blocks_sol, THREADS_PER_BLOCK>>>
			(
				d_buf_assem_sol,
				d_neighbours,
				solver_params,
				sim_params,
				boundaries,
				dt,
				dx_finest,
				test_case
			);

			CHECK_CUDA_ERROR(peek());
			CHECK_CUDA_ERROR(sync());

			rkdg2 = true;

			dg2_update<<<num_blocks_sol, THREADS_PER_BLOCK>>>
			(
				d_neighbours, 
				d_buf_assem_sol, 
				d_assem_sol, 
				solver_params, 
				sim_params, 
				dx_finest, 
				dy_finest, 
				dt, 
				test_case, 
				d_dt_CFL,
				rkdg2
			);

			CHECK_CUDA_ERROR(peek());
			CHECK_CUDA_ERROR(sync());
		}

		dt = get_dt_CFL(d_dt_CFL, d_assem_sol.length);

		CHECK_CUDA_ERROR(peek());
		CHECK_CUDA_ERROR(sync());

		// --------------------------------------------- //
		// -------------- WRITING TO FILE -------------- //
		// --------------------------------------------- //

		if ( saveint.save(time_now) )
		{
			project_assem_sol
			(
				mesh_dim,
				d_sig_details,
				d_scale_coeffs,
				d_buf_assem_sol,
				solver_params,
				d_rev_z_order,
				d_indices,
				d_assem_sol,
				d_plot_assem_sol
			);
			
			if (plot_params.row_major)
			{
				write_soln_row_major
				(
					respath,
					mesh_dim,
					d_sig_details,
					d_scale_coeffs,
					d_buf_assem_sol,
					solver_params,
					d_rev_z_order,
					d_indices,
					d_assem_sol,
					d_plot_assem_sol,
					saveint
				);
			}

			if (plot_params.vtk)
			{
				write_soln_vtk
				(
					respath,
					d_assem_sol,
					d_dt_CFL,
					dx_finest,
					dy_finest,
					sim_params,
					solver_params,
					saveint
				);
			}
			
			if (plot_params.planar)
			{
				write_soln_planar
				(
					respath,
					d_assem_sol,
					dx_finest,
					dy_finest,
					sim_params,
					solver_params,
					saveint
				);
			}

			if (plot_params.raster_out)
			{
				write_all_raster_maps
				(
					respath,
					d_plot_assem_sol,
					sim_params,
					solver_params,
					saveint,
					mesh_dim,
					dx_finest,
					first_t_step
				);
			}

			if (plot_params.c_prop)
			{
				write_c_prop_data
				(
					respath,
					start,
					solver_params,
					d_assem_sol,
					time_now,
					first_t_step
				);
			}
		}

		if ( massint.save(time_now) )
		{
			if (plot_params.cumulative)
			{
			    write_cumu_sim_time
			    (
			        start,
			        time_now,
			        respath,
			        first_t_step
			    );
			}
			
			project_assem_sol
			(
				mesh_dim,
				d_sig_details,
				d_scale_coeffs,
				d_buf_assem_sol,
				solver_params,
				d_rev_z_order,
				d_indices,
				d_assem_sol,
				d_plot_assem_sol
			);
			
			write_gauge_point_data
			(
				respath,
				mesh_dim,
				d_sig_details,
				d_scale_coeffs,
				d_buf_assem_sol,
				solver_params,
				plot_params,
				d_rev_z_order,
				d_indices,
				d_assem_sol,
				d_plot_assem_sol,
				p_finest_grid,
				gauge_points,
				time_now,
				first_t_step
			);
		}

		// --------------------------------------------- //
		// --------------------------------------------- //
		// --------------------------------------------- //

		compression = C(100.0) - C(100.0) * d_assem_sol.length / (sim_params.xsz * sim_params.ysz);

		//printf
		(
			"Elements: %d, compression: %f%%, time step: %.15f, steps: %d, sim time: %f\n", 
			d_assem_sol.length, compression, dt, ++steps, time_now
		);
		
 		first_t_step = false;
	}

	end = clock();

	run_time = (real)(end - start) / CLOCKS_PER_SEC;
	
	printf("Loop time: %f s\n", run_time);

	printf("Average time step: %f s\n", sim_params.time / steps);
	printf("Average kernel time: %f ms\n", avg_cuda_time);

	// =================== //
	// DEALLOCATING MEMORY //
	// =================== //

	CHECK_CUDA_ERROR( free_device(d_morton_codes) );
	CHECK_CUDA_ERROR( free_device(d_sorted_morton_codes) );
	CHECK_CUDA_ERROR( free_device(d_indices) );
	CHECK_CUDA_ERROR( free_device(d_rev_z_order) );
	CHECK_CUDA_ERROR( free_device(d_rev_row_major) );
	CHECK_CUDA_ERROR( free_device(d_eta_temp) );
	CHECK_CUDA_ERROR( free_device(d_sig_details) );
	CHECK_CUDA_ERROR( free_device(d_preflagged_details) );
	CHECK_CUDA_ERROR( free_device(d_norm_details) );
	CHECK_CUDA_ERROR( free_device(d_dt_CFL) );
	
	//reset();

	// =================== //

    return 0;
}

//==========================================================================================================//