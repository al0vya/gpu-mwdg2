#include "cuda_utils.cuh"

__host__
hipError_t sync()
{
	return hipDeviceSynchronize();
}

__host__
hipError_t peek()
{
	return hipPeekAtLastError();
}

__host__
hipError_t reset()
{
	return hipDeviceReset();
}

__host__
hipError_t copy
(
	void*  dst,
	void*  src,
	size_t bytes
)
{
	hipError_t error = hipMemcpy
	(
		dst,
		src,
		bytes,
		hipMemcpyDefault
	);

	return error;
}

__host__
hipError_t copy_async
(
	void*  dst,
	void*  src,
	size_t bytes
)
{
	hipError_t error = hipMemcpyAsync
	(
		dst,
		src,
		bytes,
		hipMemcpyDefault
	);

	return error;
}

__host__
void* malloc_device
(
	size_t bytes
)
{
	void* ptr;
	
	hipMalloc
	(
		&ptr,
		bytes
	);

	return ptr;
}

__host__
void* malloc_pinned
(
	size_t bytes
)
{
	void* ptr;

	hipHostMalloc
	(
		&ptr,
		bytes
	);

	return ptr;
}

__host__
hipError_t free_device
(
	void* ptr
)
{
	return (nullptr != ptr) ? hipFree(ptr) : hipSuccess;
}

__host__
hipError_t free_pinned
(
	void* ptr
)
{
	return (nullptr != ptr) ? hipHostFree(ptr) : hipSuccess;
}