#include "cuda_utils.cuh"

hipError_t sync()
{
	return hipDeviceSynchronize();
}

hipError_t peek()
{
	return hipPeekAtLastError();
}

hipError_t reset()
{
	return hipDeviceReset();
}

hipError_t copy
(
	void*  dst,
	void*  src,
	size_t bytes
)
{
	hipError_t error = hipMemcpy
	(
		dst,
		src,
		bytes,
		hipMemcpyDefault
	);

	return error;
}

hipError_t copy_async
(
	void*  dst,
	void*  src,
	size_t bytes
)
{
	hipError_t error = hipMemcpyAsync
	(
		dst,
		src,
		bytes,
		hipMemcpyDefault
	);

	return error;
}

__host__ __device__
void* malloc_device
(
	size_t bytes
)
{
	void* ptr;
	
	hipMalloc
	(
		&ptr,
		bytes
	);

	return ptr;
}

__host__ __device__
void* malloc_pinned
(
	size_t bytes
)
{
	void* ptr;

	hipHostMalloc
	(
		&ptr,
		bytes
	);

	return ptr;
}

__host__ __device__
hipError_t free_device
(
	void* ptr
)
{
	return (nullptr != ptr) ? hipFree(ptr) : hipSuccess;
}

__host__ __device__
hipError_t free_pinned
(
	void* ptr
)
{
	return (nullptr != ptr) ? hipHostFree(ptr) : hipSuccess;
}