#include "hip/hip_runtime.h"
#include "compute_error_kernel.cuh"

__global__
void compute_error_kernel
(
	real* d_computed,
	real* d_verified,
	real* d_errors,
	int   array_length
)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < array_length)
	{
		d_errors[idx] = abs( d_verified[idx] - d_computed[idx] );
	}
}