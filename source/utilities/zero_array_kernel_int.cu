#include "hip/hip_runtime.h"
#include "zero_array_kernel_real.cuh"

__global__
void zero_array_kernel_int
(
	int* d_array,
	int  num_threads
)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= num_threads) return;

	d_array[idx] = 0;
}