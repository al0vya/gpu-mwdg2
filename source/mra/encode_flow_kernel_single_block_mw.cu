#include "hip/hip_runtime.h"
#include "encode_flow_kernel_single_block_mw.cuh"

/*
 *
 * This kernel launches every refinement level starting from n = L, using 4^n threads.
 * These threads load the scale coefficients into shared memory.
 * These scale coefficients are the child coefficients of the sub-elements at n - 1.
 * At n - 1, there are 4^(n-1) sub-elements i.e. a quarter compared to 4^n.
 * Hence, after loading the scale coefficients into shared memory, only 4^(n-1) threads are kept.
 * Thereafter, each thread loads four child coefficients from shared memory to encode the parent and detail.
 * A block size that is a multiple of 4 is used to ensure enough child coefficients are loaded.
 * For the block sizes below, there is a refinement level at which only one thread block is launched:
 * 
 * Block size: 64.   Level at which only a single block is launched (LVL_SINGLE_BLOCK): 3.
 * Block size: 256.  Level at which only a single block is launched (LVL_SINGLE_BLOCK): 4.
 * Block size: 1024. Level at which only a single block is launched (LVL_SINGLE_BLOCK): 5.
 * 
 * In this scenario, the kernel is not relaunched, as a single block has enough threads for all subsequent levels.
 * Instead, there is an internal for-loop across levels, which writes the scale coefficients to shared memory.
 * The threads in the next iteration of the loop access the shared memory, which is visible to all threads within a block.
 * 
 */

__global__
void encode_flow_kernel_single_block_mw
(
	ScaleCoefficients d_scale_coeffs,
	Details           d_details,
	real*             d_norm_details,
	bool*             d_sig_details,
	bool*             d_preflagged_details,
	Maxes             maxes,
	SolverParams      solver_params,
	int               level,
	int               num_threads,
	bool              for_nghbrs
)
{
	HierarchyIndex t_idx = threadIdx.x;
	HierarchyIndex idx   = blockIdx.x * blockDim.x + t_idx;

	if (idx >= num_threads) return;
	
	real norm_detail   = C(0.0);
	real epsilon_local = solver_params.epsilon / ( 1 << (solver_params.L - level) );

	HierarchyIndex prev_lvl_idx = get_lvl_idx(level - 1);
	HierarchyIndex curr_lvl_idx = get_lvl_idx(level);
	HierarchyIndex next_lvl_idx = get_lvl_idx(level + 1);

	HierarchyIndex parent_idx = curr_lvl_idx + t_idx;
	HierarchyIndex child_idx  = next_lvl_idx + 4 * t_idx;

	bool is_sig = d_sig_details[parent_idx];

	if (is_sig)
	{
		real* eta0 = &d_scale_coeffs.eta0[child_idx];
		real* qx0 = &d_scale_coeffs.qx0[child_idx];
		real* qy0 = &d_scale_coeffs.qy0[child_idx];

		real* eta1x = &d_scale_coeffs.eta1x[child_idx];
		real* qx1x = &d_scale_coeffs.qx1x[child_idx];
		real* qy1x = &d_scale_coeffs.qy1x[child_idx];

		real* eta1y = &d_scale_coeffs.eta1y[child_idx];
		real* qx1y = &d_scale_coeffs.qx1y[child_idx];
		real* qy1y = &d_scale_coeffs.qy1y[child_idx];

		ChildScaleCoeffsMW child_coeffs =
		{
			{
				{  eta0[0],  eta0[1],  eta0[2],  eta0[3] },
				{ eta1x[0], eta1x[1], eta1x[2], eta1x[3] },
				{ eta1y[0], eta1y[1], eta1y[2], eta1y[3] }
			},
			{
				{  qx0[0],  qx0[1],  qx0[2],  qx0[3] },
				{ qx1x[0], qx1x[1], qx1x[2], qx1x[3] },
				{ qx1y[0], qx1y[1], qx1y[2], qx1y[3] }
			},
			{
				{  qy0[0],  qy0[1],  qy0[2],  qy0[3] },
				{ qy1x[0], qy1x[1], qy1x[2], qy1x[3] },
				{ qy1y[0], qy1y[1], qy1y[2], qy1y[3] }
			},
			{
				{ C(0.0), C(0.0), C(0.0), C(0.0) },
				{ C(0.0), C(0.0), C(0.0), C(0.0) },
				{ C(0.0), C(0.0), C(0.0), C(0.0) }
			}
		};

		ParentScaleCoeffsMW parent_coeffs = encode_scale_coeffs(child_coeffs);
		DetailMW            detail = (!for_nghbrs) ? encode_details(child_coeffs) : DetailMW{};

		norm_detail = detail.get_norm_detail(maxes);

		store_scale_coeffs
		(
			parent_coeffs,
			d_scale_coeffs,
			parent_idx
		);

		if (!for_nghbrs)
		{
			store_details
			(
				detail,
				d_details,
				parent_idx
			);
		}
	}

	if (!for_nghbrs)
	{
		d_norm_details[parent_idx] = norm_detail;

		d_sig_details[parent_idx] = (norm_detail >= epsilon_local) ? SIGNIFICANT : INSIGNIFICANT;

		if (d_preflagged_details[parent_idx] == SIGNIFICANT) d_sig_details[parent_idx] = SIGNIFICANT;
	}
}