#include "hip/hip_runtime.h"
#include "decoding_kernel_mw.cuh"

__global__
void decoding_kernel_mw
(
	bool*             d_sig_details,
	Details           d_details,
	ScaleCoefficients d_scale_coeffs,
	SolverParams      solver_params,
	int               level,
	int               num_threads
)
{
	HierarchyIndex t_idx = threadIdx.x;
	HierarchyIndex idx   = blockIdx.x * blockDim.x + t_idx;

	if (idx >= num_threads) return;

	typedef hipcub::BlockScan<int, THREADS_PER_BLOCK> block_scan;

	__shared__ union
	{
		typename block_scan::TempStorage temp_storage;
		HierarchyIndex parents[THREADS_PER_BLOCK];

	} shared;

	HierarchyIndex curr_lvl_idx = get_lvl_idx(level);
	HierarchyIndex next_lvl_idx = get_lvl_idx(level + 1);

	HierarchyIndex parent_idx = curr_lvl_idx + idx;

	int is_sig = d_sig_details[parent_idx];

	int thread_prefix_sum = 0;

	int num_sig_details = 0;

	block_scan(shared.temp_storage).ExclusiveSum
	(
		is_sig,
		thread_prefix_sum,
		num_sig_details
	);

	__syncthreads();

	if (is_sig) shared.parents[thread_prefix_sum] = parent_idx;

	__syncthreads();

	if (t_idx >= num_sig_details) return;

	parent_idx = shared.parents[t_idx];

	HierarchyIndex child_idx = next_lvl_idx + 4 * (parent_idx - curr_lvl_idx);
	
	PlanarCoefficients planar_coeffs;
	ScaleChildrenMW    children;
	SubDetailMW        subdetails;

	// Decoding eta
	planar_coeffs =
	{
		d_scale_coeffs.eta0[parent_idx],
		d_scale_coeffs.eta1x[parent_idx],
		d_scale_coeffs.eta1y[parent_idx],
	};

	subdetails = load_subdetails_mw
	(
		d_details.eta0,
		d_details.eta1x,
		d_details.eta1y,
		parent_idx
	);

	children = decode_scale_coeffs(planar_coeffs, subdetails);

	store_children_vector
	(
		children,
		d_scale_coeffs.eta0,
		d_scale_coeffs.eta1x,
		d_scale_coeffs.eta1y,
		child_idx
	);

	// Decoding qx
	planar_coeffs =
	{
		d_scale_coeffs.qx0[parent_idx],
		d_scale_coeffs.qx1x[parent_idx],
		d_scale_coeffs.qx1y[parent_idx],
	};

	subdetails = load_subdetails_mw
	(
		d_details.qx0,
		d_details.qx1x,
		d_details.qx1y,
		parent_idx
	);

	children = decode_scale_coeffs(planar_coeffs, subdetails);

	store_children_vector
	(
		children,
		d_scale_coeffs.qx0,
		d_scale_coeffs.qx1x,
		d_scale_coeffs.qx1y,
		child_idx
	);

	// Decoding qy
	planar_coeffs =
	{
		d_scale_coeffs.qy0[parent_idx],
		d_scale_coeffs.qy1x[parent_idx],
		d_scale_coeffs.qy1y[parent_idx],
	};

	subdetails = load_subdetails_mw
	(
		d_details.qy0,
		d_details.qy1x,
		d_details.qy1y,
		parent_idx
	);

	children = decode_scale_coeffs(planar_coeffs, subdetails);

	store_children_vector
	(
		children,
		d_scale_coeffs.qy0,
		d_scale_coeffs.qy1x,
		d_scale_coeffs.qy1y,
		child_idx
	);
}