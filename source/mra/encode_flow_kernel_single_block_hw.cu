#include "hip/hip_runtime.h"
#include "encode_flow_kernel_single_block_hw.cuh"

/*
 *
 * This kernel launches every refinement level starting from n = L, using 4^n threads.
 * These threads load the scale coefficients into shared memory.
 * These scale coefficients are the child coefficients of the sub-elements at n - 1.
 * At n - 1, there are 4^(n-1) sub-elements i.e. a quarter compared to 4^n.
 * Hence, after loading the scale coefficients into shared memory, only 4^(n-1) threads are kept.
 * Thereafter, each thread loads four child coefficients from shared memory to encode the parent and detail.
 * A block size that is a multiple of 4 is used to ensure enough child coefficients are loaded.
 * For the block sizes below, there is a refinement level at which only one thread block is launched:
 * 
 * Block size: 64.   Level at which only a single block is launched (LVL_SINGLE_BLOCK): 3.
 * Block size: 256.  Level at which only a single block is launched (LVL_SINGLE_BLOCK): 4.
 * Block size: 1024. Level at which only a single block is launched (LVL_SINGLE_BLOCK): 5.
 * 
 * In this scenario, the kernel is not relaunched, as a single block has enough threads for all subsequent levels.
 * Instead, there is an internal for-loop across levels, which writes the scale coefficients to shared memory.
 * The threads in the next iteration of the loop access the shared memory, which is visible to all threads within a block.
 * 
 */

__global__
void encode_flow_kernel_single_block_hw
(
	ScaleCoefficients d_scale_coeffs,
	Details           d_details,
	real*             d_norm_details,
	bool*             d_sig_details,
	bool*             d_preflagged_details,
	Maxes             maxes,
	SolverParams      solver_params,
	int               level,
	int               num_threads
)
{
	for (int level_kernel = level; level_kernel >= 0; level_kernel--)
	{
		const int num_threads_active = 1 << (2 * level_kernel);

		const int tidx = threadIdx.x;

		if (tidx < num_threads_active)
		{
			real norm_detail   = C(0.0);
			real epsilon_local = ( solver_params.epsilon > C(0.0) ) 
		                         ? solver_params.epsilon / ( 1 << (solver_params.L - level_kernel) )
						         : C(9999.0);
			
			const HierarchyIndex curr_lvl_idx = get_lvl_idx(level_kernel);
			const HierarchyIndex next_lvl_idx = get_lvl_idx(level_kernel + 1);
			const HierarchyIndex parent_idx   = curr_lvl_idx + tidx;
			const HierarchyIndex child_idx    = next_lvl_idx + 4 * tidx;

			ScaleChildrenHW children;
			SubDetailHW     subdetail;

			bool is_sig = d_sig_details[parent_idx];

			if (is_sig)
			{
				// Encoding eta
				load_children_vector
				(
					children,
					d_scale_coeffs.eta0,
					child_idx
				);

				d_scale_coeffs.eta0[parent_idx] = encode_scale(children);

				subdetail =
				{
					encode_detail_alpha(children),
					encode_detail_beta(children),
					encode_detail_gamma(children)
				};

				d_details.eta0.alpha[parent_idx] = subdetail.alpha;
				d_details.eta0.beta[parent_idx] = subdetail.beta;
				d_details.eta0.gamma[parent_idx] = subdetail.gamma;

				norm_detail = max(norm_detail, subdetail.get_max() / maxes.eta);

				// encoding qx
				load_children_vector
				(
					children,
					d_scale_coeffs.qx0,
					child_idx
				);

				d_scale_coeffs.qx0[parent_idx] = encode_scale(children);

				subdetail =
				{
					encode_detail_alpha(children),
					encode_detail_beta(children),
					encode_detail_gamma(children)
				};

				d_details.qx0.alpha[parent_idx] = subdetail.alpha;
				d_details.qx0.beta[parent_idx] = subdetail.beta;
				d_details.qx0.gamma[parent_idx] = subdetail.gamma;

				norm_detail = max(norm_detail, subdetail.get_max() / maxes.qx);

				// encoding qy
				load_children_vector
				(
					children,
					d_scale_coeffs.qy0,
					child_idx
				);

				d_scale_coeffs.qy0[parent_idx] = encode_scale(children);

				subdetail =
				{
					encode_detail_alpha(children),
					encode_detail_beta(children),
					encode_detail_gamma(children)
				};

				d_details.qy0.alpha[parent_idx] = subdetail.alpha;
				d_details.qy0.beta[parent_idx] = subdetail.beta;
				d_details.qy0.gamma[parent_idx] = subdetail.gamma;

				norm_detail = max(norm_detail, subdetail.get_max() / maxes.qy);

				d_norm_details[parent_idx] = norm_detail;

				d_sig_details[parent_idx] = (norm_detail >= epsilon_local || d_preflagged_details[parent_idx] == SIGNIFICANT)
					? SIGNIFICANT
					: INSIGNIFICANT;
			}
		}

		__syncthreads();
	}
}