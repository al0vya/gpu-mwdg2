#include "hip/hip_runtime.h"
#include "get_compaction_flags.cuh"

__global__
void get_compaction_flags
(
	AssembledSolution d_assem_sol,
	CompactionFlags   d_compaction_flags,
	int               num_finest_elems
)
{
	HierarchyIndex idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= num_finest_elems) return;

	HierarchyIndex current = d_assem_sol.act_idcs[idx];

	if ( idx < (num_finest_elems - 1) )
	{
		HierarchyIndex right = d_assem_sol.act_idcs[idx + 1];

		d_compaction_flags.north_east[idx] = !(current == right);
	}
	else
	{
		d_compaction_flags.north_east[idx] = 1;
	}

	if (idx > 0)
	{
		HierarchyIndex left = d_assem_sol.act_idcs[idx - 1];

		d_compaction_flags.south_west[idx] = !(current == left);
	}
	else
	{
		d_compaction_flags.south_west[idx] = 1;
	}
}