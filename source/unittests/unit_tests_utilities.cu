#include "hip/hip_runtime.h"
#include "unit_tests_utilities.cuh"

#if _RUN_UNIT_TESTS

#define TEST_MESSAGE_PASSED_ELSE_FAILED { printf("Passed %s!\n", __func__); } else { printf("Failed %s.\n", __func__); }

void unit_test_get_mean_from_array()
{
	const int array_length = 100000;
	const size_t bytes = array_length * sizeof(real);
	real* h_array = new real[array_length];
	real* d_array = (real*)malloc_device(bytes);

	for (int i = 0; i < array_length; i++)
	{
		h_array[i] = i+1;
	}

	copy_cuda(d_array, h_array, bytes);

	// sum S of 1 to n is S = n * (n+1) / 2
	// therefore, the mean M = S / n = (n+1)/2
	const real expected = (array_length + 1) / C(2.0);
	const real actual   = get_mean_from_array(d_array, array_length);

	delete[] h_array;
	free_device(d_array);

	if ( are_reals_equal( actual, expected, C(1e-2) ) )
		TEST_MESSAGE_PASSED_ELSE_FAILED
}

void unit_test_compute_error()
{
	const int array_length = 100000;
	const size_t bytes = array_length * sizeof(real);
	real* h_computed = new real[array_length];
	real* h_verified = new real[array_length];
	real* d_computed = (real*)malloc_device(bytes);
	real* d_verified = (real*)malloc_device(bytes);

	for (int i = 0; i < array_length; i++)
	{
		h_computed[i] =  i + 1;      // array is y = x
		h_verified[i] = (i + 1) * 2; // array is y = 2x
	}

	copy_cuda(d_computed, h_computed, bytes);
	copy_cuda(d_verified, h_verified, bytes);

	// array of errors = abs(d_computed - d_verified), which looks like abs(x - 2x) = x
	// i.e. the integers from 1 to 100000
	// sum S of integers 1 to n is S = n * (n+1) / 2
	// therefore, the mean M = S / n = (n+1)/2
	const real expected_error = (array_length + 1) / C(2.0);
	const real actual_error   = compute_error(d_computed, d_verified, array_length);

	delete[] h_computed;
	delete[] h_verified;
	free_device(d_computed);
	free_device(d_verified);

	if ( are_reals_equal( actual_error, expected_error, C(1e-2) ) )
		TEST_MESSAGE_PASSED_ELSE_FAILED
}

void run_unit_tests_utilities()
{
	unit_test_get_mean_from_array();
	unit_test_compute_error();
}

#endif