#include "hip/hip_runtime.h"
#include "unit_tests_utilities.cuh"

#if _RUN_UNIT_TESTS

#define TEST_MESSAGE_PASSED_ELSE_FAILED { printf("Passed %s!\n", __func__); } else { printf("Failed %s.\n", __func__); }

void unit_test_get_max_from_array()
{
	const int array_length = 100000;
	const size_t bytes = array_length * sizeof(real);
	real* h_array = new real[array_length];
	real* d_array = (real*)malloc_device(bytes);

	for (int i = 0; i < array_length; i++)
	{
		h_array[i] = i+1;
	}

	copy_cuda(d_array, h_array, bytes);

	const real expected = array_length;
	const real actual   = get_max_from_array(d_array, array_length);

	delete[] h_array;
	free_device(d_array);

	if ( are_reals_equal( actual, expected, C(1e-2) ) )
		TEST_MESSAGE_PASSED_ELSE_FAILED
}

void unit_test_get_mean_from_array()
{
	const int array_length = 100000;
	const size_t bytes = array_length * sizeof(real);
	real* h_array = new real[array_length];
	real* d_array = (real*)malloc_device(bytes);

	for (int i = 0; i < array_length; i++)
	{
		h_array[i] = i+1;
	}

	copy_cuda(d_array, h_array, bytes);

	// sum S of 1 to n is S = n * (n+1) / 2
	// therefore, the mean M = S / n = (n+1)/2
	const real expected = (array_length + 1) / C(2.0);
	const real actual   = get_mean_from_array(d_array, array_length);

	delete[] h_array;
	free_device(d_array);

	if ( are_reals_equal( actual, expected, C(1e-2) ) )
		TEST_MESSAGE_PASSED_ELSE_FAILED
}

void unit_test_compute_error()
{
	const int array_length = 100000;
	const size_t bytes = array_length * sizeof(real);
	real* h_computed = new real[array_length];
	real* h_verified = new real[array_length];
	real* d_computed = (real*)malloc_device(bytes);
	real* d_verified = (real*)malloc_device(bytes);

	for (int i = 0; i < array_length; i++)
	{
		h_computed[i] =  i + 1;      // array is y = x
		h_verified[i] = (i + 1) * 2; // array is y = 2x
	}

	copy_cuda(d_computed, h_computed, bytes);
	copy_cuda(d_verified, h_verified, bytes);

	// array of errors = abs(d_computed - d_verified), which looks like abs(x - 2x) = x
	// i.e. the integers from 1 to 100000
	// sum S of integers 1 to n is S = n * (n+1) / 2
	// therefore, the mean M = S / n = (n+1)/2
	const real expected_error = array_length;
	const real actual_error   = compute_error(d_computed, d_verified, array_length);

	delete[] h_computed;
	delete[] h_verified;
	free_device(d_computed);
	free_device(d_verified);

	if ( are_reals_equal( actual_error, expected_error, C(1e-2) ) )
		TEST_MESSAGE_PASSED_ELSE_FAILED
}

void unit_test_compare_array_on_device_vs_host_real()
{
	const int array_length = 100;
	const size_t bytes = array_length * sizeof(real);
	real* h_array = new real[array_length];
	real* d_array = (real*)malloc_device(bytes);

	for (int i = 0; i < array_length; i++)
	{
		h_array[i] = i;
	}

	copy_cuda(d_array, h_array, bytes);

	const real actual_error   = compare_array_on_device_vs_host_real(h_array, d_array, array_length);
	const real expected_error = C(1e-6);

	delete[] h_array;
	free_device(d_array);

	if ( are_reals_equal(actual_error, expected_error) )
		TEST_MESSAGE_PASSED_ELSE_FAILED
}

void unit_test_compare_array_with_file_bool()
{
	const int array_length = 100;
	bool* h_array = new bool[array_length];

	for (int i = 0; i < array_length; i++)
	{
		h_array[i] = i % 2 == 0;
	}

	const char* dirroot  = "unittestdata";
	const char* filename = "unit_test_compare_array_with_file_bool";

	const int differences = compare_array_with_file_bool(dirroot, filename, h_array, array_length);

	delete[] h_array;

	if (differences == 0)
		TEST_MESSAGE_PASSED_ELSE_FAILED
}

void unit_test_compare_array_with_file_real()
{
	const int array_length = 100;
	real* h_array = new real[array_length];

	for (int i = 0; i < array_length; i++)
	{
		h_array[i] = i;
	}

	const char* dirroot  = "unittestdata";
	const char* filename = "unit_test_compare_array_with_file_real";

	const real actual_error   = compare_array_with_file_real(dirroot, filename, h_array, array_length);
	const real expected_error = C(1e-6);

	delete[] h_array;

	if ( are_reals_equal(actual_error, expected_error) )
		TEST_MESSAGE_PASSED_ELSE_FAILED
}

void unit_test_compare_d_array_with_file_bool()
{
	const int array_length = 100;
	const size_t bytes = array_length * sizeof(bool);
	bool* h_array = new bool[array_length];
	bool* d_array = (bool*)malloc_device(bytes);

	for (int i = 0; i < array_length; i++)
	{
		h_array[i] = i % 2 == 0;
	}
	
	const char* dirroot  = "unittestdata";
	const char* filename = "unit_test_compare_d_array_with_file_bool";

	copy_cuda(d_array, h_array, bytes);

	const int differences = compare_d_array_with_file_bool(dirroot, filename, d_array, array_length);

	delete[] h_array;
	free_device(d_array);

	if (differences == 0)
		TEST_MESSAGE_PASSED_ELSE_FAILED
}

void unit_test_compare_d_array_with_file_real()
{
	const int array_length = 100;
	const size_t bytes = array_length * sizeof(real);
	real* h_array = new real[array_length];
	real* d_array = (real*)malloc_device(bytes);

	for (int i = 0; i < array_length; i++)
	{
		h_array[i] = i;
	}
	
	const char* dirroot  = "unittestdata";
	const char* filename = "unit_test_compare_d_array_with_file_real";

	copy_cuda(d_array, h_array, bytes);

	const real actual_error   = compare_d_array_with_file_real(dirroot, filename, d_array, array_length);
	const real expected_error = C(1e-6);

	delete[] h_array;
	free_device(d_array);

	if ( are_reals_equal(actual_error, expected_error) )
		TEST_MESSAGE_PASSED_ELSE_FAILED
}

void run_unit_tests_utilities()
{
	unit_test_get_max_from_array();
	unit_test_get_mean_from_array();
	unit_test_compute_error();
	unit_test_compare_array_on_device_vs_host_real();
	unit_test_compare_array_with_file_bool();
	unit_test_compare_array_with_file_real();
	unit_test_compare_d_array_with_file_bool();
	unit_test_compare_d_array_with_file_real();
}

#endif